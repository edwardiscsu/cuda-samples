
#include "hip/hip_runtime.h"


#include <stdio.h>

#define MAXBLOCKS 1
#define MAXTHREADS 10

//Helper method
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

//__global__ (paralellized method)
__global__ void VectorAdd(int *c, const int *a, const int *b)
{
    int i = threadIdx.x; //Assign each c element to a single thread
	c[i] = a[i] + b[i];
}

int main()
{
	int *a, *b, *c;      //CPU

	//Allocate CPU memory
	a = (int*)malloc(MAXTHREADS*sizeof(int));
	b = (int*)malloc(MAXTHREADS*sizeof(int));
	c = (int*)malloc(MAXTHREADS*sizeof(int));

	for (int i = 0; i < MAXTHREADS; ++i) //Populate array
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

    //Call "surrogate" method
	hipError_t cudaStatus = addWithCuda(c, a, b, MAXTHREADS);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	//Display result
    printf("\nMAXTHREADS (%d) VECTOR ADDITION USING CUDA\n\n", MAXTHREADS);
	printf("c[i] = a[i] + b[i]\n");
	printf("======================================\n");
	for (int i = 0; i < MAXTHREADS; ++i)
		printf("a[%d] = %d, b[%d] = %d, c[%d] = %d\n", i, a[i], i, b[i], i, c[i]);

	//Free CPU memory
	free(a);
	free(b);
	free(c);

    //cudaDeviceReset must be called before exiting in order for profiling and
    //tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

//Helper/"surrogate" method for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *d_a = 0;//GPU
    int *d_b = 0;//GPU
    int *d_c = 0;//GPU
    hipError_t cudaStatus;

    //Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    //Allocate GPU memory
    cudaStatus = hipMalloc((void**)&d_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&d_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&d_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    //Copy data to GPU
    cudaStatus = hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	//Run GPU using MAXBLOCK number of blocks and size number of threads
	VectorAdd<<<MAXBLOCKS, size>>>(d_c, d_a, d_b); 

    //Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    //cudaDeviceSynchronize waits for the kernel to finish, and returns
    //any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    //Copy result back to CPU
    cudaStatus = hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
	//Free GPU memory
    hipFree(d_c);
    hipFree(d_a);
    hipFree(d_b);
    
    return cudaStatus;
}
