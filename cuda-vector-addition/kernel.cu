
#include "hip/hip_runtime.h"


#include <stdio.h>

#define SIZE 10

void VectorAdd(int *a, int *b, int*c, int n)
{
	int i;
	for (i = 0; i < n; ++i)
		c[i] = a[i] + b[i];
}

int main()
{
	int *a, *b, *c;

	a = (int*)malloc(SIZE*sizeof(int));
	b = (int*)malloc(SIZE*sizeof(int));
	c = (int*)malloc(SIZE*sizeof(int));

	for (int i = 0; i < SIZE; ++i) //Populate array
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	VectorAdd(a, b, c, SIZE);

	for (int i = 0; i < 10; ++i)
		printf("c[%d] = %d\n", i, c[i]);

	free(a);
	free(b);
	free(c);
	
	return 0;
}