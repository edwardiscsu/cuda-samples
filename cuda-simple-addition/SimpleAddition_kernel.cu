
#include "hip/hip_runtime.h"


#include <stdio.h>

#define MAXBLOCKS 1
#define MAXTHREADS 1

__global__ void SimpleAddition(int *a, int *b, int *c)
{
    *c = *a + *b;
}

int main()
{
    int a, b, c;         //CPU
	int *d_a, *d_b, *d_c;//GPU

	//Allocate GPU memory
	hipMalloc((void **)&d_a, sizeof(int));
	hipMalloc((void **)&d_b, sizeof(int));
	hipMalloc((void **)&d_c, sizeof(int));

	a = 1;
	b = 2;
	c = 0;

	//Copy data to GPU
	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, &c, sizeof(int), hipMemcpyHostToDevice);

	SimpleAddition<<<MAXBLOCKS, MAXTHREADS>>>(d_a, d_b, d_c);

	//Copy result back to CPU
	hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

	printf("%d + %d = %d\n", a, b, c);

	//Free GPU memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

    return 0;
}